#include "CubStorageBuffer.cuh"
#include <hipcub/hipcub.hpp>

CubStorageBuffer::CubStorageBuffer()
{
	storage = NULL;
	size = 0;
}

CubStorageBuffer::~CubStorageBuffer()
{
	hipFree(&storage);
	hipFree(&size);
}

CubStorageBuffer CubStorageBuffer::AllocateCubSortBuffer(VolumeEdepPair edepPairList, uint64_t nVals)
{
	//Create the buffer with default constructor
	CubStorageBuffer returnBuffer = CubStorageBuffer();

	//Call the CUB function to determine memory constraints, then malloc
	hipcub::DeviceRadixSort::SortPairs(returnBuffer.storage,returnBuffer.size,edepPairList.volume,edepPairList.volume,edepPairList.edep,edepPairList.edep,nVals);
	hipMalloc(&returnBuffer.storage,returnBuffer.size); 

	return returnBuffer;
}

CubStorageBuffer CubStorageBuffer::AllocateCubReduceBuffer(VolumeEdepPair edepPairList, uint64_t nVals)
{
	//Create the buffer with default constructor
	CubStorageBuffer returnBuffer = CubStorageBuffer();
	//Generic Reduction Operator
	CUBAddOperator reductionOperator;

	//Call the CUB function to determine memory constraints, then malloc
	hipcub::DeviceReduce::ReduceByKey(returnBuffer.storage,returnBuffer.size, edepPairList.volume, edepPairList.volume, edepPairList.edep, edepPairList.edep, edepPairList.numElements, reductionOperator, nVals);
	hipMalloc(&returnBuffer.storage,returnBuffer.size); 

	return returnBuffer;
}

CubStorageBuffer CubStorageBuffer::AllocateCubHistogramBuffer(VolumeEdepPair edepPairList, uint64_t nVals, int* histogramVals, double* logBins, int nbins)
{
	//Create the buffer with default constructor
	CubStorageBuffer returnBuffer = CubStorageBuffer();

	//Call the CUB function to determine memory constraints, then malloc
	hipcub::DeviceHistogram::HistogramRange(returnBuffer.storage,returnBuffer.size, edepPairList.edep,histogramVals,nbins+1,logBins,nVals);
	hipMalloc(&returnBuffer.storage,returnBuffer.size); 

	return returnBuffer;
}