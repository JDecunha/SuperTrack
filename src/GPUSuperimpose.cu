#include "hip/hip_runtime.h"
//SuperTrack
#include "testCUDA.cuh"
#include "utils.hh"
#include "SuperTrackTypes.cuh"
//ROOT
#include "TROOT.h"
#include "TFile.h"
#include "TH1F.h"
#include "TEntryList.h"
#include "TTree.h"
#include "TTreeReader.h"
#include "TMath.h"
#include "ROOT/TProcessExecutor.hxx"
//STD
#include <vector>
#include <iterator>
#include <tuple>
#include <filesystem>
//CUDA Libraries
#include <hip/hip_runtime.h>
#include <hiprand.h>
//CUB (Cuda UnBound)
#include <hipcub/hipcub.hpp>


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void LoadTrack(const std::tuple<Int_t,Int_t,Int_t,TString> &input, Track *hostTrack, Track *deviceTrack)
{
	//Open the file in each process and make a Tree Reader
	TFile f = TFile(std::get<3>(input));
	TTreeReader trackReader("Tracks", &f);
	trackReader.SetEntriesRange(std::get<0>(input),std::get<1>(input));
	TTreeReaderValue<double_t> xReader(trackReader, "x [nm]");
	TTreeReaderValue<double_t> yReader(trackReader, "y [nm]");
	TTreeReaderValue<double_t> zReader(trackReader, "z [nm]");
	TTreeReaderValue<double_t> edepReader(trackReader, "edep [eV]");

	std::cout << "thread #: " << std::get<2>(input) << " starting at: " << std::to_string(std::get<0>(input)) << std::endl;

	//Determine size of arrays. Define them. Then allcate unified memory on CPU and GPU
	long nVals = std::get<1>(input) - std::get<0>(input) + 1; //+1 because number of values includes first and last value
	size_t trackSize = nVals * sizeof(double);
	size_t trackStructSize = nVals*sizeof(Track);

	//malloc and hipMalloc our arrays respectively
	hostTrack->x = (double *)malloc(trackSize);
	hostTrack->y = (double *)malloc(trackSize);
	hostTrack->z = (double *)malloc(trackSize);
	hostTrack->edep = (double *)malloc(trackSize);
	hipMalloc(&(deviceTrack->x),trackSize);
	hipMalloc(&(deviceTrack->y),trackSize);
	hipMalloc(&(deviceTrack->z),trackSize);
	hipMalloc(&(deviceTrack->edep),trackSize);

	//Fill the unified memory arrays from the CPU
	for (long loopnum = 0; trackReader.Next(); loopnum++) 
	{
		hostTrack->x[loopnum] = *xReader;
		hostTrack->y[loopnum] = *yReader;
		hostTrack->z[loopnum] = *zReader;
		hostTrack->edep[loopnum] = *edepReader;
	}

	//Copy track to GPU memory
	hipMemcpy(deviceTrack->x,hostTrack->x,trackSize,hipMemcpyHostToDevice);
	hipMemcpy(deviceTrack->y,hostTrack->y,trackSize,hipMemcpyHostToDevice);
	hipMemcpy(deviceTrack->z,hostTrack->z,trackSize,hipMemcpyHostToDevice);
	hipMemcpy(deviceTrack->edep,hostTrack->edep,trackSize,hipMemcpyHostToDevice);

	//TODO: free host track, maybe we can't free it yet because it's still being copied right
}

void GenerateRandomXYShift(const std::tuple<Int_t,Int_t,Int_t,TString> &input, float **randomVals, const int &nSamples, const long &random_seed)
{
	hipMalloc(randomVals,2*sizeof(float)*nSamples); 
	
	//Random number generation on GPU
	hiprandGenerator_t randGenerator;
	hiprandCreateGenerator(&randGenerator,HIPRAND_RNG_PSEUDO_DEFAULT); //consider changing this to Mersenne Twister later
	hiprandSetPseudoRandomGeneratorSeed(randGenerator,random_seed+std::get<2>(input));
	hiprandGenerateUniform(randGenerator,*randomVals,2*nSamples);
	hiprandDestroyGenerator(randGenerator);
	hipDeviceSynchronize();
}

void GenerateLogHistogram(double **logBins, int **histogramVals, int **histogramValsAccumulated, int nbins, float binLowerMagnitude, float binUpperMagnitude)
{
	//Get the device Id for active GPU
	int deviceId;
	hipGetDevice(&deviceId);   

	//Fill the log bins and send to the device
	hipMallocManaged(logBins, (nbins+1)*sizeof(double));
	LogSpace(binLowerMagnitude,binUpperMagnitude,nbins,*logBins);
	hipMemPrefetchAsync(*logBins,(nbins+1)*sizeof(double),deviceId);

	//TODO: Change to unmanaged memory later
	hipMallocManaged(histogramVals,nbins*sizeof(int));
	hipMallocManaged(histogramValsAccumulated,nbins*sizeof(int));

	//Set arrays to zero
	hipMemset(*histogramVals,0,nbins*sizeof(int));
	hipMemset(*histogramValsAccumulated,0,nbins*sizeof(int));
}

__global__ void FilterInScoringBox(SphericalGeometry geometry, float* randomVals, Track inputTrack, Track outputTrack, int numElements, int *numElementsCompacted, int oversampleIterationNumber)
{
	//This function
	//1.) Applies the random shift to the x,y coordinates
	//2.) Checks which edep events are in the box
	//3.) Performs stream compaction on those events which are in the box
	//We are using stream compaction to avoid a monolithic kernel with large blocks within if-statements which reduces warp efficiency

	//Put definitions outside of for-loop to prevent repeat constructor calls
	double x_shifted; double y_shifted; int outputIndex;

	//Determine index and strid
  int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	//Counters for shared memory atomics
	int localPosition;
	__shared__ int localIndexCounter;
	
	//Convert random shifts in to appropriate range
	double x_shift = ((randomVals[(oversampleIterationNumber*2)]*geometry.greatestSphereOffset*2)-geometry.greatestSphereOffset);
	double y_shift = ((randomVals[(oversampleIterationNumber*2+1)]*geometry.greatestSphereOffset*2)-geometry.greatestSphereOffset);

	//The value we compare to, to check if it's in the box
	double box_edge = abs(geometry.greatestSphereOffset)+(geometry.sphereRadius);

	//Loop over all the energy deposition points
	for (int i = index; i < numElements; i+=stride)
	{
		//Apply random shift
		x_shifted = inputTrack.x[i] + x_shift;
		y_shifted = inputTrack.y[i] + y_shift;

		//Set local position to negative value, only takes on positive value if predicate is true
		localPosition = -1;

		//Zero the local counter
		if (threadIdx.x == 0) 
		{
			localIndexCounter = 0;
		}
		__syncthreads();

		//Check if in box, if true assign the local index position
		if (abs(x_shifted) < box_edge  && abs(y_shifted) < box_edge)
		{
			localPosition = atomicAdd(&localIndexCounter,1);
		}
		__syncthreads();

		//Add the local counter to the global counter
		if (threadIdx.x == 0)
		{
			localIndexCounter = atomicAdd(numElementsCompacted,localIndexCounter);
		}
		__syncthreads();

		//If predicate is true, then write the track to position localCounter+localPosition (localCounter now stores the globalCounter value because of the atomic add)
		if(localPosition != -1)
		{
			//Atomically add to the global counter for the output array length
			outputIndex = localPosition+localIndexCounter;
			//printf("current loop i: %d",i);
			//Copy the track inside the box over to the new array
			outputTrack.x[outputIndex] = x_shifted;
			outputTrack.y[outputIndex] = y_shifted;
			outputTrack.z[outputIndex] = inputTrack.z[i];
			outputTrack.edep[outputIndex] = inputTrack.edep[i];
		}
		__syncthreads();
	}
}

__global__ void FilterTrackInSphere(SphericalGeometry geometry, Track inputTrack, int *numElements, int *numElementsCompacted, int *trackIdInSphere)
{

	//printf("%f %f \n",geometry.sphereDiameter,geometry.scoringRegionHalfLength);

	//move all of the variable definitions out of the for loop
	double distFromNearestSphereX, distFromNearestSphereY, distFromNearestSphereZ, dist;

	//Determine index and stride
 	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	//Counters for shared memory atomics
	int localPosition;
	__shared__ int localIndexCounter;

	//Pre-calculate values
	double sphereDiameter = geometry.sphereDiameter; 
	double sphereRadiusMag = geometry.sphereRadius*geometry.sphereRadius; 

	//Loop over all the energy deposition points
	for (long i = index; i < *numElements; i+=stride)
	{		
		//Find distance to the nearest sphere.
		//For performance reasons, we work in an arbitrary coordinate system here, rather than the "global" coordinate system
		//The "global" coordinate systrem is relative to the greatest sphere offset
		//In the later scoring kernel we work in the global coordinate system, and that's why we subtract the greatest sphere offset there
		distFromNearestSphereX = llrint((inputTrack.x[i])/sphereDiameter)*geometry.sphereDiameter-(inputTrack.x[i]);
		distFromNearestSphereY = llrint((inputTrack.y[i])/sphereDiameter)*geometry.sphereDiameter-(inputTrack.y[i]); 
		distFromNearestSphereZ = llrint((inputTrack.z[i])/sphereDiameter)*geometry.sphereDiameter-(inputTrack.z[i]); 

		//Determine if inside the nearest sphere
		dist = (distFromNearestSphereX*distFromNearestSphereX)+(distFromNearestSphereY*distFromNearestSphereY)+(distFromNearestSphereZ*distFromNearestSphereZ);

		//Set local position to negative value, only takes on positive value if predicate is true
		localPosition = -1;

		//Zero the local counter
		if (threadIdx.x == 0) 
		{
			localIndexCounter = 0;
		}
		__syncthreads();

		//Check if in sphere, then assign local index position
		if (dist <= sphereRadiusMag)
		{
			localPosition = atomicAdd(&localIndexCounter,1);
			
		}
		__syncthreads();

		//Add the local counter to the global counter
		if (threadIdx.x == 0)
		{
			localIndexCounter = atomicAdd(numElementsCompacted,localIndexCounter);
		}
		__syncthreads();

		//If predicate is true, then write the track to position localCounter+localPosition (localCounter now stores the globalCounter value because of the atomic add)
		if (localPosition != -1)
		{
			//Atomically add to the global counter for the output array length
			trackIdInSphere[localPosition+localIndexCounter] = i;
		}
		__syncthreads();

	}
}

__global__ void ScoreTrackInSphere(SphericalGeometry geometry, Track inputTrack, int *numElements, int *trackIdInSphere, VolumeEdepPair outputPair)
{
	//move all of the variable definitions out of the for loop
	long xIndex, yIndex, zIndex, sphereHitIndex;

	//Pre-calculate some values
	double sphereDiameter = geometry.sphereDiameter;
	double linealDenominator = (2./3.)*sphereDiameter; 

	//Determine index and stride
 	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	//Loop over all the energy deposition points
	for (uint64_t i = index; i < *numElements; i+=stride)
	{		
		xIndex = llrint((inputTrack.x[trackIdInSphere[i]]-geometry.greatestSphereOffset)/sphereDiameter);
		yIndex = llrint((inputTrack.y[trackIdInSphere[i]]-geometry.greatestSphereOffset)/sphereDiameter);
		zIndex = llrint((inputTrack.z[trackIdInSphere[i]]-geometry.greatestSphereOffset)/sphereDiameter);

		//Determine the Index of the sphere hit
		sphereHitIndex = xIndex + yIndex*geometry.numSpheresLinear+ zIndex*geometry.numSpheresLinear*geometry.numSpheresLinear; //Keep in mind that for the index it starts counting at zero

		//Write to volumeID and edepOutput
		outputPair.volume[i] = sphereHitIndex;
		outputPair.edep[i] = inputTrack.edep[trackIdInSphere[i]]/linealDenominator; //this should be ev/nm which is same a kev/um
	}
}

struct CUBAddOperator
{
    template <typename T>
    HIPCUB_RUNTIME_FUNCTION __forceinline__
    T operator()(const T &a, const T &b) const {
        return a+b;
    }
};

__global__ void AccumulateHistogramVals(int* temp, int* accumulated,int N)
{
	//Determine index and stride
 	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i < N; i+=stride)
	{
		accumulated[i] = accumulated[i]+temp[i];
	}
}

//TODO:Change this back to Malloc after testing
VolumeEdepPair AllocateGPUVolumeEdepPair(uint64_t numElements)
{
	VolumeEdepPair toAllocate;
	hipMalloc(&(toAllocate.volume),numElements*sizeof(uint64_t));
	hipMalloc(&(toAllocate.edep),numElements*sizeof(double));
	hipMallocManaged(&(toAllocate.numElements),sizeof(int));

	return toAllocate;
}

//Todo: create function for freeing GPUVolumeEdepPair

CubStorageBuffer AllocateCubSortBuffer(VolumeEdepPair edepPairList, uint64_t nVals)
{
	//Create the buffer with default constructor
	CubStorageBuffer returnBuffer = CubStorageBuffer();

	//Call the CUB function to determine memory constraints, then malloc
	hipcub::DeviceRadixSort::SortPairs(returnBuffer.storage,returnBuffer.size,edepPairList.volume,edepPairList.volume,edepPairList.edep,edepPairList.edep,nVals);
	hipMalloc(&returnBuffer.storage,returnBuffer.size); 

	return returnBuffer;
}

CubStorageBuffer AllocateCubReduceBuffer(VolumeEdepPair edepPairList, uint64_t nVals)
{
	//Create the buffer with default constructor
	CubStorageBuffer returnBuffer = CubStorageBuffer();
	//Generic Reduction Operator
	CUBAddOperator reductionOperator;

	//Call the CUB function to determine memory constraints, then malloc
	hipcub::DeviceReduce::ReduceByKey(returnBuffer.storage,returnBuffer.size, edepPairList.volume, edepPairList.volume, edepPairList.edep, edepPairList.edep, edepPairList.numElements, reductionOperator, nVals);
	hipMalloc(&returnBuffer.storage,returnBuffer.size); 

	return returnBuffer;
}

CubStorageBuffer AllocateCubHistogramBuffer(VolumeEdepPair edepPairList, uint64_t nVals, int* histogramVals, double* logBins, int nbins)
{
	//Create the buffer with default constructor
	CubStorageBuffer returnBuffer = CubStorageBuffer();

	//Call the CUB function to determine memory constraints, then malloc
	hipcub::DeviceHistogram::HistogramRange(returnBuffer.storage,returnBuffer.size, edepPairList.edep,histogramVals,nbins+1,logBins,nVals);
	hipMalloc(&returnBuffer.storage,returnBuffer.size); 

	return returnBuffer;
}

__global__ void SortReduceHistogram(CubStorageBuffer sortBuffer, CubStorageBuffer reduceBuffer, CubStorageBuffer histogramBuffer, VolumeEdepPair edepsInTarget, VolumeEdepPair sortedEdeps, VolumeEdepPair reducedEdeps, int nbins,int* histogramVals, double* logBins, CUBAddOperator reductionOperator)
{
	//Sort the edep volume pairs
	hipcub::DeviceRadixSort::SortPairs(sortBuffer.storage,sortBuffer.size,edepsInTarget.volume,sortedEdeps.volume,edepsInTarget.edep,sortedEdeps.edep,*(edepsInTarget.numElements));
	// reduce the energy depositions
	hipcub::DeviceReduce::ReduceByKey(reduceBuffer.storage,reduceBuffer.size, sortedEdeps.volume, reducedEdeps.volume, sortedEdeps.edep, reducedEdeps.edep, reducedEdeps.numElements, reductionOperator, *(edepsInTarget.numElements));
	//Create the histogram
	hipcub::DeviceHistogram::HistogramRange(histogramBuffer.storage,histogramBuffer.size,reducedEdeps.edep,histogramVals,nbins+1,logBins,*reducedEdeps.numElements);
}

__global__ void ZeroInt(int* toZero)
{
	*toZero = 0;
}

TH1F score_lineal_GPU(TString filepath, float_t scoring_sphere_spacing, float_t scoring_sphere_diameter, Int_t nthreads, Int_t nSamples = 20000, Long_t random_seed = time(NULL))
{
	//open the file and retrieve the trees
	TFile f = TFile(filepath);
	TTree *trackIndex;
	f.GetObject("Track index",trackIndex);
	long long nTracksInFile = trackIndex->GetEntries();

	//Populate our tuple with the first entry, last entry, and random seed for each thread
	std::vector<std::tuple<Int_t,Int_t,Int_t,TString>> perthread_input_arguments;

	if (nTracksInFile <= nthreads)
	{ 
		long start_entry_val = 0;
		TTreeReader trackIndexReader("Track index", &f);
		TTreeReaderValue<long long> end_entry_val(trackIndexReader, "index");

		for (Int_t i = 0; i < nTracksInFile; i++)
		{
			trackIndexReader.Next();
			perthread_input_arguments.push_back(std::make_tuple(start_entry_val,*end_entry_val-1,i,filepath));
			//Wcout << "thread: " << i << " start val: " << start_entry_val << " end val: " << *end_entry_val-1 << endl;
			start_entry_val = *end_entry_val;
		}
	}
	else
	{
		long start_entry_val = 0;
		TTreeReader trackIndexReader("Track index", &f);
		TTreeReaderValue<long long> end_entry_val(trackIndexReader, "index");

		for (Int_t i = 0; i < nthreads; i++)
		{
			trackIndexReader.Next();
			perthread_input_arguments.push_back(std::make_tuple(start_entry_val,*end_entry_val-1,i,filepath));
			//Wcout << "thread: " << i << " start val: " << start_entry_val << " end val: " << *end_entry_val-1 << endl;
			start_entry_val = *end_entry_val;
		}
		std::cout << "Number of tracks in file greater than requested threads. Case not yet implemented." << std::endl;
	}

	//Pull my geometry information to get passed to each of my threads
	TNamed* voxelSideLength;
	f.GetObject("Voxel side length [mm]",voxelSideLength);
	float scoring_square_half_length = atof(voxelSideLength->GetTitle())*1e6;
	SphericalGeometry sphericalGeometry = SphericalGeometry(scoring_square_half_length,scoring_sphere_diameter);

	//We are done reading the Tree single threaded. Close it.
	f.Close();

	auto workItem = [=](std::tuple<Int_t,Int_t,Int_t,TString> input) //the = sign captures everything in the enclosing function by value. Meaning it makes a process local copy.
	{
		//Calculate size information for memory allocations
		int nVals = std::get<1>(input) - std::get<0>(input) + 1; //+1 because number of values includes first and last value
		size_t trackSize = nVals * sizeof(double); 
		size_t trackStructSize = nVals *sizeof(Track);

		//Define local and GPU track pointers
		Track hostTrack; Track deviceTrack; 
		LoadTrack(input, &hostTrack, &deviceTrack); //load track from disk and copy to GPU

		//Allocate memory for the tracks found to be within the box
		Track randomlyShiftedTrack; 
		hipMalloc(&(randomlyShiftedTrack.x),trackSize); 
		hipMalloc(&(randomlyShiftedTrack.y),trackSize); 
		hipMalloc(&(randomlyShiftedTrack.z),trackSize); 
		hipMalloc(&(randomlyShiftedTrack.edep),trackSize); 

		//Allocate memory to store the TrackIDs of the points within spheres
		int *inSphereTrackId;
		hipMalloc(&inSphereTrackId,nVals*sizeof(int));
		
		//Allocate GPU only memory for random numbers
		float *randomVals; 
		GenerateRandomXYShift(input, &randomVals, nSamples, random_seed); //Allocate and fill with random numbers

		//Define histograms
		double *logBins; 
		int* histogramVals; 
		int* histogramValsAccumulated; 
		int nbins = 200; float binLowerMagnitude = -1; float binUpperMagnitude = 2; //Set histogram parameters
		GenerateLogHistogram(&logBins, &histogramVals, &histogramValsAccumulated, nbins, binLowerMagnitude, binUpperMagnitude);
		
		//Allocate GPU only memory for the volume:edep paired list
		VolumeEdepPair edepsInTarget = AllocateGPUVolumeEdepPair(nVals);
		VolumeEdepPair sortedEdeps = AllocateGPUVolumeEdepPair(nVals); 
		VolumeEdepPair reducedEdeps = AllocateGPUVolumeEdepPair(nVals); 

		int *NumInBox; 
		hipMallocManaged(&NumInBox,sizeof(int)); 

		CUBAddOperator reductionOperator;

		//Allocate memory for the temporary storage the CUB operations needs
		CubStorageBuffer sortBuffer = AllocateCubSortBuffer(edepsInTarget,nVals);
		CubStorageBuffer reduceBuffer = AllocateCubReduceBuffer(edepsInTarget,nVals);
		CubStorageBuffer histogramBuffer = AllocateCubHistogramBuffer(edepsInTarget,nVals,histogramVals,logBins,nbins);

		//Configure cuda kernel launches
		/*int blockSize;
		int minGridSize;
		int gridSize;
		hipOccupancyMaxPotentialBlockSize(&minGridSize,&blockSize,ScoreTrackInSphere,0,0);
		gridSize = (nVals + blockSize - 1)/blockSize;
		std::cout << gridSize << " " << blockSize << std::endl;*/

		for (int j = 0; j < nSamples; j++)
		{

			//New track. Zero values
			ZeroInt<<<1,1>>>(NumInBox);
			ZeroInt<<<1,1>>>(edepsInTarget.numElements);

			//Filter and score the tracks
			FilterInScoringBox<<<256,256>>>(sphericalGeometry,randomVals,deviceTrack,randomlyShiftedTrack,nVals,NumInBox,j);	
			FilterTrackInSphere<<<256,256>>>(sphericalGeometry,randomlyShiftedTrack,NumInBox,edepsInTarget.numElements,inSphereTrackId); 
			ScoreTrackInSphere<<<256,256>>>(sphericalGeometry,randomlyShiftedTrack,edepsInTarget.numElements,inSphereTrackId,edepsInTarget); 

			//Sort the edeps by volumeID, reduce (accumulate), and then place into histograms
			SortReduceHistogram<<<1,1>>>(sortBuffer,reduceBuffer,histogramBuffer,edepsInTarget,sortedEdeps,reducedEdeps, nbins,histogramVals,logBins, reductionOperator);

			//Accumulate the histogram values
			AccumulateHistogramVals<<<4,32>>>(histogramVals,histogramValsAccumulated,nbins);
		}

		int number_of_values_in_histogram = 0;
		hipDeviceSynchronize();
		//Read out histogram
		for (int i = 0; i < nbins; i++)
		{
			number_of_values_in_histogram += histogramValsAccumulated[i];
			std::cout << "Bin: " << logBins[i] << " Counts: " << histogramValsAccumulated[i] << std::endl;
		}

		std::cout << number_of_values_in_histogram << std::endl;
		//TODO: close my file at some point */
	  
	  //Initialize the histogram
		TH1F lineal_histogram = TH1F("Lineal energy histogram", "y*f(y)", 200, -2,1);
		return lineal_histogram;

		//TODO: Free all the memory I allocated too
		hipFree(inSphereTrackId);
		hipFree(randomVals);


	};


	// Create the pool of workers
	ROOT::TProcessExecutor workers(nthreads);
	//Process the jobs and get a vector of the output
	std::vector<TH1F> process_output = workers.Map(workItem, perthread_input_arguments);

	TH1F lineal_histogram = TH1F("Lineal energy histogram", "y*f(y)", 200, -2,1);

	return lineal_histogram;

}