#include "hip/hip_runtime.h"
//SuperTrack
#include "testCUDA.cuh"
#include "utils.hh"
//ROOT
#include "TROOT.h"
#include "TFile.h"
#include "TH1F.h"
#include "TEntryList.h"
#include "TTree.h"
#include "TTreeReader.h"
#include "TMath.h"
#include "TCanvas.h"
#include "THStack.h"
#include "TPad.h"
#include "ROOT/TProcessExecutor.hxx"
//STD
#include <unordered_map>
#include <vector>
#include <iterator>
#include <tuple>
#include <filesystem>
//CUDA Libraries
#include <hip/hip_runtime.h>
#include <hiprand.h>

TH1F score_lineal_GPU(TString filepath, float_t scoring_sphere_spacing, float_t scoring_sphere_diameter, Int_t nthreads, Int_t nSamples = 1, Long_t random_seed = time(NULL))
{
	//open the file and retrieve the trees
	TFile f = TFile(filepath);
	TTree *trackIndex;
	f.GetObject("Track index",trackIndex);
	long long nTracksInFile = trackIndex->GetEntries();

	//Populate our tuple with the first entry, last entry, and random seed for each thread
	std::vector<std::tuple<Int_t,Int_t,Int_t,TString>> perthread_input_arguments;

	//TODO: update the GEANT code to use long long for the event index
	if (nTracksInFile <= nthreads)
	{ 
		long start_entry_val = 0;
		TTreeReader trackIndexReader("Track index", &f);
		TTreeReaderValue<long long> end_entry_val(trackIndexReader, "index");

		for (Int_t i = 0; i < nTracksInFile; i++)
		{
			trackIndexReader.Next();
			perthread_input_arguments.push_back(std::make_tuple(start_entry_val,*end_entry_val-1,i,filepath));
			//Wcout << "thread: " << i << " start val: " << start_entry_val << " end val: " << *end_entry_val-1 << endl;
			start_entry_val = *end_entry_val;
		}
	}
	else
	{
		std::cout << "Number of tracks in file greater than requested threads. Case not yet implemented." << std::endl;
	}

	//We are done reading the Tree single threaded. Close it.
	f.Close();


	
	//the = sign captures everything in the enclosing function by value. Meaning it makes a process local copy.
	auto workItem = [=](std::tuple<Int_t,Int_t,Int_t,TString> input) 
	{
		//Open the file in each process and make a Tree Reader
		TFile f = TFile(std::get<3>(input));
		TTreeReader trackReader("Tracks", &f);
		trackReader.SetEntriesRange(std::get<0>(input),std::get<1>(input));
		TTreeReaderValue<double_t> xReader(trackReader, "x [nm]");
		TTreeReaderValue<double_t> yReader(trackReader, "y [nm]");
		TTreeReaderValue<double_t> zReader(trackReader, "z [nm]");
		TTreeReaderValue<double_t> edepReader(trackReader, "edep [eV]");

		std::cout << "thread #: " << std::get<2>(input) << " starting at: " << std::to_string(std::get<0>(input)) << std::endl;

		//Determine size of arrays. Define them. Then allcate unified memory on CPU and GPU
		long nVals = std::get<1>(input) - std::get<0>(input) + 1; //+1 because number of values includes first and last value
		size_t trackSize = nVals * sizeof(double);

		double *x;
		double *y;
		double *z;
		double *edep;

		hipMallocManaged(&x,trackSize);
		hipMallocManaged(&y,trackSize);
		hipMallocManaged(&z,trackSize);
		hipMallocManaged(&edep,trackSize);

		//Fill the unified memory arrays from the CPU
		for (long loopnum = 0; trackReader.Next(); loopnum++) 
		{
			x[loopnum] = *xReader;
			y[loopnum] = *yReader;
			z[loopnum] = *zReader;
			edep[loopnum] = *edepReader;
		}

		//Get the device Id for active GPU
		int deviceId;
		hipGetDevice(&deviceId);    
		//Prefetch memory by the GPU
		hipMemPrefetchAsync(x,trackSize,deviceId);
		hipMemPrefetchAsync(y,trackSize,deviceId);
		hipMemPrefetchAsync(z,trackSize,deviceId);
		hipMemPrefetchAsync(edep,trackSize,deviceId);

		//Allocate GPU only memory for the random numbers
		float *randomVals;
		hipMalloc(&randomVals,2*sizeof(float)*nSamples); //2 values for x,y times the number of oversamples needed
		
		//Random number generation on GPU
		hiprandGenerator_t randGenerator;
		hiprandCreateGenerator(&randGenerator,HIPRAND_RNG_PSEUDO_DEFAULT); //consider changing this to Mersenne Twister later
		hiprandSetPseudoRandomGeneratorSeed(randGenerator,random_seed+std::get<2>(input));
		hiprandGenerateUniform(randGenerator,randomVals,2*nSamples);
		hiprandDestroyGenerator(randGenerator);

		//Allocate GPU only memory for the volume:edep paired list
		long *volumeID;
		double *edepInVolume;

		hipMalloc(&volumeID,sizeof(long)*nVals);
		hipMalloc(&edepInVolume,trackSize);

		//TODO: Invoke superimposing kernel call here

		//TODO: Consolidate results of superimposing into a hash table

		//TODO: Transform hash table into a histogram

		//TODO: Transfer histogram back to CPU memory and return

		//


	  	//Initialize the histogram
		TH1F lineal_histogram = TH1F("Lineal energy histogram", "y*f(y)", 200, -2,1);
		return lineal_histogram;

	};


	// Create the pool of workers
	ROOT::TProcessExecutor workers(nthreads);
	//Process the jobs and get a vector of the output
	std::vector<TH1F> process_output = workers.Map(workItem, perthread_input_arguments);

	TH1F lineal_histogram = TH1F("Lineal energy histogram", "y*f(y)", 200, -2,1);

	return lineal_histogram;

}

//TODO: Change this to work with a C-style struct later, so x,y,z,edep are all one entry
__global__ void SuperimposeTrack(double greatestSphereOffset, double sphereDiameter, long numSpheresLinear, double* randomVals, double* x, double* y, double* z, double* edep,long *volumeID, double *edepOutput, long numElements)
{
	//Our entire geometry should be able to be described by only the greatest offset, the sphere diameter and number of spheres in a line. That's useful
	double sphereRadius = sphereDiameter/2;
	double linealDenominator = (2./3.)*sphereDiameter; //calculate this here as an efficiency gain

	//Determine index and stride
 	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	//Loop over all the energy deposition points
	for (long i = index; i < numElements; i+=stride)
	{
		//Write a zero to edepOutput and volumeID. Doing this here avoids warp divergence later.
		edepOutput[i] = 0; volumeID[i] = 0;

		//Apply random shift
		double x_shifted = x[i] + randomVals[(i*2)];
		double y_shifted = y[i] + randomVals[(i*2)+1];

		//Determine inside box
		if (abs(x_shifted) < abs(greatestSphereOffset)+(sphereRadius) && abs(y_shifted) < abs(greatestSphereOffset)+(sphereRadius) && abs(z[i]) < abs(greatestSphereOffset)+(sphereRadius))
		{
			//Convert position to index in the grid of spheres
			long xIndex = llround((x_shifted-greatestSphereOffset)/sphereDiameter);
			long yIndex = llround((y_shifted-greatestSphereOffset)/sphereDiameter);
			long zIndex = llround((z[i]-greatestSphereOffset)/sphereDiameter);
			
			//Determine the location of the nearest sphere in the grid (with 0,0,0 being the top left sphere, different coordinate system than the ptcls are in)
			double nearestSphereX = xIndex*sphereDiameter;
			double nearestSphereY = yIndex*sphereDiameter;
			double nearestSphereZ = zIndex*sphereDiameter;

			//Find the distance from the nearest sphere. You have to shift x_shift by gSO to get in the same coordinate system as the sphere grid
			//An aside: I feel like there is probably a way that you could define the sphere grid that might reduce the complexity of this kernel
			//Another aside: calculating in cubes would reduce complexity as well
			double distFromNearestSphereX = nearestSphereX-(x_shifted-greatestSphereOffset);
			double distFromNearestSphereY = nearestSphereY-(y_shifted-greatestSphereOffset); 
			double distFromNearestSphereZ = nearestSphereZ-(z[i]-greatestSphereOffset); 

			//Determine if inside the nearest sphere
			double dist = pow(distFromNearestSphereX,2)+pow(distFromNearestSphereY,2)+pow(distFromNearestSphereZ,2);
			dist = sqrt(dist);

			if (dist <= sphereRadius)
			{
				//Determine the Index of the sphere hit
				long sphereHitIndex = xIndex + yIndex*(numSpheresLinear) + zIndex*pow(numSpheresLinear,2); //Keep in mind that for the index it starts counting at zero

				//Write to volumeID and edepOutput
				volumeID[i] = sphereHitIndex;
				edepOutput[i] = edep[i]/linealDenominator; //this should be ev/nm which is same a kev/um
			}
		}
	}


}