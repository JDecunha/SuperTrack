#include "hip/hip_runtime.h"
//SuperTrack
#include "testCUDA.cuh"
#include "utils.hh"
#include "SuperTrackTypes.cuh"
//ROOT
#include "TROOT.h"
#include "TFile.h"
#include "TH1F.h"
#include "TEntryList.h"
#include "TTree.h"
#include "TTreeReader.h"
#include "TMath.h"
#include "ROOT/TProcessExecutor.hxx"
//STD
#include <vector>
#include <iterator>
#include <tuple>
#include <filesystem>
//CUDA Libraries
#include <hip/hip_runtime.h>
#include <hiprand.h>
//CUB (Cuda UnBound)
#include <hipcub/hipcub.hpp>


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void LoadTrack(const std::tuple<Int_t,Int_t,Int_t,TString> &input, Track **hostTrack, Track **deviceTrack)
{
	//Open the file in each process and make a Tree Reader
	TFile f = TFile(std::get<3>(input));
	TTreeReader trackReader("Tracks", &f);
	trackReader.SetEntriesRange(std::get<0>(input),std::get<1>(input));
	TTreeReaderValue<double_t> xReader(trackReader, "x [nm]");
	TTreeReaderValue<double_t> yReader(trackReader, "y [nm]");
	TTreeReaderValue<double_t> zReader(trackReader, "z [nm]");
	TTreeReaderValue<double_t> edepReader(trackReader, "edep [eV]");

	std::cout << "thread #: " << std::get<2>(input) << " starting at: " << std::to_string(std::get<0>(input)) << std::endl;

	//Determine size of arrays. Define them. Then allcate unified memory on CPU and GPU
	long nVals = std::get<1>(input) - std::get<0>(input) + 1; //+1 because number of values includes first and last value
	size_t trackSize = nVals * sizeof(double);
	size_t trackStructSize = nVals*sizeof(Track);

	//malloc and hipMalloc our arrays respectively
	*hostTrack = (Track *)malloc(trackStructSize);
	hipMalloc(deviceTrack,trackStructSize);

	//Fill the unified memory arrays from the CPU
	for (long loopnum = 0; trackReader.Next(); loopnum++) 
	{
		(*hostTrack)[loopnum].x = *xReader;
		(*hostTrack)[loopnum].y = *yReader;
		(*hostTrack)[loopnum].z = *zReader;
		(*hostTrack)[loopnum].edep = *edepReader;
	}

	//Copy track to GPU memory
	hipMemcpy(*deviceTrack,*hostTrack,trackStructSize,hipMemcpyHostToDevice);

	//TODO: free host track, maybe we can't free it yet because it's still being copied right
}

void GenerateRandomXYShift(const std::tuple<Int_t,Int_t,Int_t,TString> &input, float **randomVals, const int &nSamples, const long &random_seed)
{
	hipMalloc(randomVals,2*sizeof(float)*nSamples); 
	
	//Random number generation on GPU
	hiprandGenerator_t randGenerator;
	hiprandCreateGenerator(&randGenerator,HIPRAND_RNG_PSEUDO_DEFAULT); //consider changing this to Mersenne Twister later
	hiprandSetPseudoRandomGeneratorSeed(randGenerator,random_seed+std::get<2>(input));
	hiprandGenerateUniform(randGenerator,*randomVals,2*nSamples);
	hiprandDestroyGenerator(randGenerator);
	hipDeviceSynchronize();
}

void GenerateLogHistogram(double **logBins, int **histogramVals, int **histogramValsAccumulated, int nbins, float binLowerMagnitude, float binUpperMagnitude)
{
	//Get the device Id for active GPU
	int deviceId;
	hipGetDevice(&deviceId);   

	//Fill the log bins and send to the device
	hipMallocManaged(logBins, (nbins+1)*sizeof(double));
	LogSpace(binLowerMagnitude,binUpperMagnitude,nbins,*logBins);
	hipMemPrefetchAsync(*logBins,(nbins+1)*sizeof(double),deviceId);

	//TODO: Change to unmanaged memory later
	hipMallocManaged(histogramVals,nbins*sizeof(int));
	hipMallocManaged(histogramValsAccumulated,nbins*sizeof(int));

	//Set arrays to zero
	hipMemset(*histogramVals,0,nbins*sizeof(int));
	hipMemset(*histogramValsAccumulated,0,nbins*sizeof(int));
}

__global__ void FilterInScoringBox(SphericalGeometry geometry, float* randomVals, Track *inputTrack, Track *outputTrack, int numElements, int *numElementsCompacted, int oversampleIterationNumber)
{
	//This function
	//1.) Applies the random shift to the x,y coordinates
	//2.) Checks which edep events are in the box
	//3.) Performs stream compaction on those events which are in the box
	//We are using stream compaction to avoid a monolithic kernel with large blocks within if-statements which reduces warp efficiency

	//Put definitions outside of for-loop to prevent repeat constructor calls
	double x_shifted; double y_shifted; int outputIndex;

	//Determine index and strid
  int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	//Counters for shared memory atomics
	int localPosition;
	__shared__ int localIndexCounter;
	
	//Convert random shifts in to appropriate range
	double x_shift = ((randomVals[(oversampleIterationNumber*2)]*geometry.greatestSphereOffset*2)-geometry.greatestSphereOffset);
	double y_shift = ((randomVals[(oversampleIterationNumber*2+1)]*geometry.greatestSphereOffset*2)-geometry.greatestSphereOffset);

	//The value we compare to, to check if it's in the box
	double box_edge = abs(geometry.greatestSphereOffset)+(geometry.sphereRadius);

	//Loop over all the energy deposition points
	for (int i = index; i < numElements; i+=stride)
	{
		//Apply random shift
		x_shifted = inputTrack[i].x + x_shift;
		y_shifted = inputTrack[i].y + y_shift;

		//Set local position to negative value, only takes on positive value if predicate is true
		localPosition = -1;

		//Zero the local counter
		if (threadIdx.x == 0) 
		{
			localIndexCounter = 0;
		}
		__syncthreads();

		//Check if in box, if true assign the local index position
		if (abs(x_shifted) < box_edge  && abs(y_shifted) < box_edge)
		{
			localPosition = atomicAdd(&localIndexCounter,1);
		}
		__syncthreads();

		//Add the local counter to the global counter
		if (threadIdx.x == 0)
		{
			localIndexCounter = atomicAdd(numElementsCompacted,localIndexCounter);
		}
		__syncthreads();

		//If predicate is true, then write the track to position localCounter+localPosition (localCounter now stores the globalCounter value because of the atomic add)
		if(localPosition != -1)
		{
			//Atomically add to the global counter for the output array length
			outputIndex = localPosition+localIndexCounter;
			//printf("current loop i: %d",i);
			//Copy the track inside the box over to the new array
			outputTrack[outputIndex].x = x_shifted;
			outputTrack[outputIndex].y = y_shifted;
			outputTrack[outputIndex].z = inputTrack[i].z;
			outputTrack[outputIndex].edep = inputTrack[i].edep;
		}
		__syncthreads();
	}
}

__global__ void FilterTrackInSphere(SphericalGeometry geometry, Track *inputTrack, int *numElements, int *numElementsCompacted, int *trackIdInSphere)
{

	//printf("%f %f \n",geometry.sphereDiameter,geometry.scoringRegionHalfLength);

	//move all of the variable definitions out of the for loop
	double distFromNearestSphereX, distFromNearestSphereY, distFromNearestSphereZ, dist;

	//Determine index and stride
 	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	//Counters for shared memory atomics
	int localPosition;
	__shared__ int localIndexCounter;

	//Pre-calculate values
	double sphereDiameter = geometry.sphereDiameter; 
	double sphereRadiusMag = geometry.sphereRadius*geometry.sphereRadius; 

	//Loop over all the energy deposition points
	for (long i = index; i < *numElements; i+=stride)
	{		
		//Find distance to the nearest sphere.
		//For performance reasons, we work in an arbitrary coordinate system here, rather than the "global" coordinate system
		//The "global" coordinate systrem is relative to the greatest sphere offset
		//In the later scoring kernel we work in the global coordinate system, and that's why we subtract the greatest sphere offset there
		distFromNearestSphereX = llrint((inputTrack[i].x)/sphereDiameter)*geometry.sphereDiameter-(inputTrack[i].x);
		distFromNearestSphereY = llrint((inputTrack[i].y)/sphereDiameter)*geometry.sphereDiameter-(inputTrack[i].y); 
		distFromNearestSphereZ = llrint((inputTrack[i].z)/sphereDiameter)*geometry.sphereDiameter-(inputTrack[i].z); 

		//Determine if inside the nearest sphere
		dist = (distFromNearestSphereX*distFromNearestSphereX)+(distFromNearestSphereY*distFromNearestSphereY)+(distFromNearestSphereZ*distFromNearestSphereZ);

		//Set local position to negative value, only takes on positive value if predicate is true
		localPosition = -1;

		//Zero the local counter
		if (threadIdx.x == 0) 
		{
			localIndexCounter = 0;
		}
		__syncthreads();

		//Check if in sphere, then assign local index position
		if (dist <= sphereRadiusMag)
		{
			localPosition = atomicAdd(&localIndexCounter,1);
			
		}
		__syncthreads();

		//Add the local counter to the global counter
		if (threadIdx.x == 0)
		{
			localIndexCounter = atomicAdd(numElementsCompacted,localIndexCounter);
		}
		__syncthreads();

		//If predicate is true, then write the track to position localCounter+localPosition (localCounter now stores the globalCounter value because of the atomic add)
		if (localPosition != -1)
		{
			//Atomically add to the global counter for the output array length
			trackIdInSphere[localPosition+localIndexCounter] = i;
		}
		__syncthreads();

	}
}

__global__ void ScoreTrackInSphere(SphericalGeometry geometry, Track *inputTrack, int *numElements, int *trackIdInSphere, VolumeEdepPair outputPair)
{
	//move all of the variable definitions out of the for loop
	long xIndex, yIndex, zIndex, sphereHitIndex;

	//Pre-calculate some values
	double sphereDiameter = geometry.sphereDiameter;
	double linealDenominator = (2./3.)*sphereDiameter; 

	//Determine index and stride
 	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	//Loop over all the energy deposition points
	for (uint64_t i = index; i < *numElements; i+=stride)
	{		
		xIndex = llrint((inputTrack[trackIdInSphere[i]].x-geometry.greatestSphereOffset)/sphereDiameter);
		yIndex = llrint((inputTrack[trackIdInSphere[i]].y-geometry.greatestSphereOffset)/sphereDiameter);
		zIndex = llrint((inputTrack[trackIdInSphere[i]].z-geometry.greatestSphereOffset)/sphereDiameter);

		//Determine the Index of the sphere hit
		sphereHitIndex = xIndex + yIndex*(geometry.numSpheresLinear) + zIndex*pow(geometry.numSpheresLinear,2); //Keep in mind that for the index it starts counting at zero

		//Write to volumeID and edepOutput
		outputPair.volume[i] = sphereHitIndex;
		outputPair.edep[i] = inputTrack[trackIdInSphere[i]].edep/linealDenominator; //this should be ev/nm which is same a kev/um
	}
}

struct CUBAddOperator
{
    template <typename T>
    HIPCUB_RUNTIME_FUNCTION __forceinline__
    T operator()(const T &a, const T &b) const {
        return a+b;
    }
};

__global__ void AccumulateHistogramVals(int* temp, int* accumulated,int N)
{
	//Determine index and stride
 	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i < N; i+=stride)
	{
		accumulated[i] = accumulated[i]+temp[i];
	}
}
__global__ void NonsenseKernel(float *value)
{
	printf("Took in the value:%f", value[0]);
	/*int lastval = 0;
	for (int i = 0; i < 10; i++)
	{
		lastval = atomicAdd(atomicValue,1);
		printf("Last atomic value %d: \n",lastval);
	}*/
}

void testfunction(hiprandGenerator_t &gen,float *vals)
{
	hiprandGenerateUniform(gen,vals,2*20000);
	hipDeviceSynchronize();
}

void mallocfunction(float **vals)
{
	//okay wait, so calling &vals in here, is the pointer to the local object, and not the address of vals
	//Wow! so I pass in a pointer to a pointer. And then feed that to hipMalloc direclty.
	//Intuitively what this means it memory address pointing to pointer of array of floats --> memory address point to array of floats --> start of floats
	//We had to do this because of the way the malloc works. In c you can't really pass by reference, so when you're mallocing something
	//you need to pass a pointer, to the pointer that you then want to work on
	//I promise it makes sense if you think about it for a moment.

	hipMalloc(vals,2*sizeof(float)*20000); 
	hipDeviceSynchronize();
}

void readHostTrack(Track *hostTrack)
{
	for(int i = 0; i < 100000; i++)
	{
		std::cout << hostTrack[i].x << std::endl;
	}
}
__global__ void readDeviceTrack(Track *deviceTrack)
{
	for(int i = 0;i<100000;i++)
	{
		printf("%f \n",deviceTrack[i].x);
	}
}
__global__ void readDeviceEdepList(double *edeps)
{
	for(int i = 0;i<200;i++)
	{
		printf("%f \n",edeps[i]);
	}
}

__global__ void indexTestingKernel()
{
 	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	printf("index %d, stride %d \n",index,stride);
}

/*__global__ void extractCudaInformation()
{
	printf("Cuda_ARCH: %s",__CUDA_ARCH__);
	printf("CUDA_ACC: %s",__HIPCC__);
	printf("CUDA_RDC: %s",__CUDACC_RDC__);
}*/

//TODO:Change this back to Malloc after testing
VolumeEdepPair AllocateGPUVolumeEdepPair(uint64_t numElements)
{
	VolumeEdepPair toAllocate;
	hipMalloc(&(toAllocate.volume),numElements*sizeof(uint64_t));
	hipMalloc(&(toAllocate.edep),numElements*sizeof(double));
	hipMallocManaged(&(toAllocate.numElements),sizeof(int));

	return toAllocate;
}

//Todo: create function for freeing GPUVolumeEdepPair

CubStorageBuffer AllocateCubSortBuffer(VolumeEdepPair edepPairList, uint64_t nVals)
{
	//Create the buffer with default constructor
	CubStorageBuffer returnBuffer = CubStorageBuffer();

	//Call the CUB function to determine memory constraints, then malloc
	hipcub::DeviceRadixSort::SortPairs(returnBuffer.storage,returnBuffer.size,edepPairList.volume,edepPairList.volume,edepPairList.edep,edepPairList.edep,nVals);
	hipMalloc(&returnBuffer.storage,returnBuffer.size); 

	return returnBuffer;
}

CubStorageBuffer AllocateCubReduceBuffer(VolumeEdepPair edepPairList, uint64_t nVals)
{
	//Create the buffer with default constructor
	CubStorageBuffer returnBuffer = CubStorageBuffer();
	//Generic Reduction Operator
	CUBAddOperator reductionOperator;

	//Call the CUB function to determine memory constraints, then malloc
	hipcub::DeviceReduce::ReduceByKey(returnBuffer.storage,returnBuffer.size, edepPairList.volume, edepPairList.volume, edepPairList.edep, edepPairList.edep, edepPairList.numElements, reductionOperator, nVals);
	hipMalloc(&returnBuffer.storage,returnBuffer.size); 

	return returnBuffer;
}

CubStorageBuffer AllocateCubHistogramBuffer(VolumeEdepPair edepPairList, uint64_t nVals, int* histogramVals, double* logBins, int nbins)
{
	//Create the buffer with default constructor
	CubStorageBuffer returnBuffer = CubStorageBuffer();

	//Call the CUB function to determine memory constraints, then malloc
	hipcub::DeviceHistogram::HistogramRange(returnBuffer.storage,returnBuffer.size, edepPairList.edep,histogramVals,nbins+1,logBins,nVals);
	hipMalloc(&returnBuffer.storage,returnBuffer.size); 

	return returnBuffer;
}

__global__ void ReadVolumeEdepPair(VolumeEdepPair* pair)
{
	for (int i = 0; i < 10000; i++)
	{
		printf("volume: %d",pair->volume[i]);
	}	
}

TH1F score_lineal_GPU(TString filepath, float_t scoring_sphere_spacing, float_t scoring_sphere_diameter, Int_t nthreads, Int_t nSamples = 20000, Long_t random_seed = time(NULL))
{
	//open the file and retrieve the trees
	TFile f = TFile(filepath);
	TTree *trackIndex;
	f.GetObject("Track index",trackIndex);
	long long nTracksInFile = trackIndex->GetEntries();

	//Populate our tuple with the first entry, last entry, and random seed for each thread
	std::vector<std::tuple<Int_t,Int_t,Int_t,TString>> perthread_input_arguments;

	if (nTracksInFile <= nthreads)
	{ 
		long start_entry_val = 0;
		TTreeReader trackIndexReader("Track index", &f);
		TTreeReaderValue<long long> end_entry_val(trackIndexReader, "index");

		for (Int_t i = 0; i < nTracksInFile; i++)
		{
			trackIndexReader.Next();
			perthread_input_arguments.push_back(std::make_tuple(start_entry_val,*end_entry_val-1,i,filepath));
			//Wcout << "thread: " << i << " start val: " << start_entry_val << " end val: " << *end_entry_val-1 << endl;
			start_entry_val = *end_entry_val;
		}
	}
	else
	{
		long start_entry_val = 0;
		TTreeReader trackIndexReader("Track index", &f);
		TTreeReaderValue<long long> end_entry_val(trackIndexReader, "index");

		for (Int_t i = 0; i < nthreads; i++)
		{
			trackIndexReader.Next();
			perthread_input_arguments.push_back(std::make_tuple(start_entry_val,*end_entry_val-1,i,filepath));
			//Wcout << "thread: " << i << " start val: " << start_entry_val << " end val: " << *end_entry_val-1 << endl;
			start_entry_val = *end_entry_val;
		}
		std::cout << "Number of tracks in file greater than requested threads. Case not yet implemented." << std::endl;
	}

	//Pull my geometry information to get passed to each of my threads
	TNamed* voxelSideLength;
	f.GetObject("Voxel side length [mm]",voxelSideLength);
	float scoring_square_half_length = atof(voxelSideLength->GetTitle())*1e6;
	int num_spheres_linear = TMath::Ceil(((scoring_square_half_length*2)/scoring_sphere_spacing)); 
	long long int num_spheres_total = TMath::Power((num_spheres_linear),3);
	double_t top_sphere_offset = -(((float(num_spheres_linear))/2)-0.5)*scoring_sphere_spacing;
	float_t scoringSphereRadius = scoring_sphere_diameter/2;

	SphericalGeometry sphericalGeometry = SphericalGeometry(scoring_square_half_length,scoring_sphere_diameter);

	//We are done reading the Tree single threaded. Close it.
	f.Close();

	auto workItem = [=](std::tuple<Int_t,Int_t,Int_t,TString> input) //the = sign captures everything in the enclosing function by value. Meaning it makes a process local copy.
	{
		//Calculate size information for memory allocations
		int nVals = std::get<1>(input) - std::get<0>(input) + 1; //+1 because number of values includes first and last value
		size_t trackSize = nVals * sizeof(double); 
		size_t trackStructSize = nVals *sizeof(Track);

		//Define local and GPU track pointers
		Track *hostTrack; Track *deviceTrack; 
		LoadTrack(input, &hostTrack, &deviceTrack); //load track from disk and copy to GPU

		//Allocate memory for the tracks found to be within the box
		Track *randomlyShiftedTrack; 
		hipMalloc(&randomlyShiftedTrack,trackStructSize); 

		//Allocate memory to store the TrackIDs of the points within spheres
		int *inSphereTrackId;
		hipMalloc(&inSphereTrackId,nVals*sizeof(int));
		
		//Allocate GPU only memory for random numbers
		float *randomVals; 
		GenerateRandomXYShift(input, &randomVals, nSamples, random_seed); //Allocate and fill with random numbers

		//Define histograms
		double *logBins; 
		int* histogramVals; 
		int* histogramValsAccumulated; 
		int nbins = 200; float binLowerMagnitude = -1; float binUpperMagnitude = 2; //Set histogram parameters
		GenerateLogHistogram(&logBins, &histogramVals, &histogramValsAccumulated, nbins, binLowerMagnitude, binUpperMagnitude);
		
		//Allocate GPU only memory for the volume:edep paired list
		VolumeEdepPair edepsInTarget = AllocateGPUVolumeEdepPair(nVals);
		VolumeEdepPair sortedEdeps = AllocateGPUVolumeEdepPair(nVals); 
		VolumeEdepPair reducedEdeps = AllocateGPUVolumeEdepPair(nVals); 

		int *NumInBox; 
		hipMallocManaged(&NumInBox,sizeof(int)); 

		CUBAddOperator reductionOperator;

		//Allocate memory for the temporary storage the CUB operations needs
		CubStorageBuffer sortBuffer = AllocateCubSortBuffer(edepsInTarget,nVals);
		CubStorageBuffer reduceBuffer = AllocateCubReduceBuffer(edepsInTarget,nVals);
		CubStorageBuffer histogramBuffer = AllocateCubHistogramBuffer(edepsInTarget,nVals,histogramVals,logBins,nbins);


		for (int j = 0; j < nSamples; j++)
		{

			//New track. Zero values
			hipMemset(NumInBox,0,sizeof(int));
			hipMemset(edepsInTarget.numElements,0,sizeof(int));

			FilterInScoringBox<<<60,256>>>(sphericalGeometry,randomVals,deviceTrack,randomlyShiftedTrack,nVals,NumInBox,j);	
			FilterTrackInSphere<<<60,256>>>(sphericalGeometry,randomlyShiftedTrack,NumInBox,edepsInTarget.numElements,inSphereTrackId); 
			ScoreTrackInSphere<<<60,256>>>(sphericalGeometry,randomlyShiftedTrack,edepsInTarget.numElements,inSphereTrackId,edepsInTarget); 
		
			hipDeviceSynchronize();

			//I think, because the sort_by_key operation takes *NumInSpheres as an argument
			//If the kernel call is given, before NumInSpheres has finished updating, then it gets an incorrect value

			//Sort the edep volume pairs
			hipcub::DeviceRadixSort::SortPairs(sortBuffer.storage,sortBuffer.size,edepsInTarget.volume,sortedEdeps.volume,edepsInTarget.edep,sortedEdeps.edep,*(edepsInTarget.numElements));

			// reduce the energy depositions
			hipcub::DeviceReduce::ReduceByKey(reduceBuffer.storage,reduceBuffer.size, sortedEdeps.volume, reducedEdeps.volume, sortedEdeps.edep, reducedEdeps.edep, reducedEdeps.numElements, reductionOperator, *(edepsInTarget.numElements));

			//Create the histogram
			hipcub::DeviceHistogram::HistogramRange(histogramBuffer.storage,histogramBuffer.size,reducedEdeps.edep,histogramVals,nbins+1,logBins,*reducedEdeps.numElements);

			//Accumulate the histogram values
			AccumulateHistogramVals<<<4,32>>>(histogramVals,histogramValsAccumulated,nbins);
	
		}

		int number_of_values_in_histogram = 0;
		hipDeviceSynchronize();
		//Read out histogram
		for (int i = 0; i < nbins; i++)
		{
			number_of_values_in_histogram += histogramValsAccumulated[i];
			std::cout << "Bin: " << logBins[i] << " Counts: " << histogramValsAccumulated[i] << std::endl;
		}

		std::cout << number_of_values_in_histogram << std::endl;
		//TODO: close my file at some point */
		//hipDeviceSynchronize();
	  	//Initialize the histogram
		TH1F lineal_histogram = TH1F("Lineal energy histogram", "y*f(y)", 200, -2,1);
		return lineal_histogram;

	};


	// Create the pool of workers
	ROOT::TProcessExecutor workers(nthreads);
	//Process the jobs and get a vector of the output
	std::vector<TH1F> process_output = workers.Map(workItem, perthread_input_arguments);

	TH1F lineal_histogram = TH1F("Lineal energy histogram", "y*f(y)", 200, -2,1);

	return lineal_histogram;

}

//TODO: Change this to work with a C-style struct later, so x,y,z,edep are all one entry
/*__global__ void SuperimposeTrack(double greatestSphereOffset, double sphereDiameter, long numSpheresLinear, float* randomVals, double* x, double* y, double* z, double* edep,long *volumeID, double *edepOutput, long numElements,int oversampleIterationNumber)
{
	//Our entire geometry should be able to be described by only the greatest offset, the sphere diameter and number of spheres in a line. That's useful
	double sphereRadius = sphereDiameter/2;
	double linealDenominator = (2./3.)*sphereDiameter; //calculate this here as an efficiency gain

	//Determine index and stride
 	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	//Convert random shifts  in to appropriate range
	double x_shift = ((randomVals[(oversampleIterationNumber*2)]*greatestSphereOffset*2)-greatestSphereOffset);
	double y_shift = ((randomVals[(oversampleIterationNumber*2+1)]*greatestSphereOffset*2)-greatestSphereOffset);
	//printf("x_shift: %f \n",randomVals[(oversampleIterationNumber*2)]);
	//printf("y_shift: %f \n",randomVals[(oversampleIterationNumber*2+1)]);
	//printf("Greatest sphere offset: %f \n", greatestSphereOffset);

	//Loop over all the energy deposition points
	for (long i = index; i < numElements; i+=stride)
	{
		//Write a zero to edepOutput and volumeID. Doing this here avoids warp divergence later.
		edepOutput[i] = 0; volumeID[i] = 0;

		//Apply random shift. My numbers that come in are floats from 0.0 to 1.0. Have to shift them to the desired range
		double x_shifted = x[i] + x_shift;
		double y_shifted = y[i] + y_shift;

		//Check if inside box
		if (abs(x_shifted) < abs(greatestSphereOffset)+(sphereRadius) && abs(y_shifted) < abs(greatestSphereOffset)+(sphereRadius) && abs(z[i]) < abs(greatestSphereOffset)+(sphereRadius))
		{
			//Convert position to index in the grid of spheres
			//printf("x_shifted: %f \n",x_shifted);
			long xIndex = llround((x_shifted-greatestSphereOffset)/sphereDiameter);
			long yIndex = llround((y_shifted-greatestSphereOffset)/sphereDiameter);
			long zIndex = llround((z[i]-greatestSphereOffset)/sphereDiameter);
			
			//Determine the location of the nearest sphere in the grid (with 0,0,0 being the top left sphere, different coordinate system than the ptcls are in)
			double nearestSphereX = xIndex*sphereDiameter;
			double nearestSphereY = yIndex*sphereDiameter;
			double nearestSphereZ = zIndex*sphereDiameter;

			//Find the distance from the nearest sphere. You have to shift x_shift by gSO to get in the same coordinate system as the sphere grid
			//An aside: I feel like there is probably a way that you could define the sphere grid that might reduce the complexity of this kernel
			//Another aside: calculating in cubes would reduce complexity as well
			double distFromNearestSphereX = nearestSphereX-(x_shifted-greatestSphereOffset);
			double distFromNearestSphereY = nearestSphereY-(y_shifted-greatestSphereOffset); 
			double distFromNearestSphereZ = nearestSphereZ-(z[i]-greatestSphereOffset); 

			//Determine if inside the nearest sphere
			double dist = pow(distFromNearestSphereX,2)+pow(distFromNearestSphereY,2)+pow(distFromNearestSphereZ,2);
			dist = sqrt(dist);

			if (dist <= sphereRadius)
			{
				//Determine the Index of the sphere hit
				long sphereHitIndex = xIndex + yIndex*(numSpheresLinear) + zIndex*pow(numSpheresLinear,2); //Keep in mind that for the index it starts counting at zero

				//Write to volumeID and edepOutput
				volumeID[i] = sphereHitIndex;
				edepOutput[i] = edep[i]/linealDenominator; //this should be ev/nm which is same a kev/um
				//printf("volumeID: %ld %ld %ld edep: %f. \n",xIndex,yIndex,zIndex,edepOutput[i]);
			}
		}
	}
}*/

/*__global__ void ScoreTrackInSphere(double greatestSphereOffset, double sphereRadius, long numSpheresLinear, Track *inputTrack, int *numElements, long *volumeID, double *edepOutput, int *numElementsCompacted)
{

	double sphereDiameter = sphereRadius*2;
	double linealDenominator = (2./3.)*sphereDiameter; 

	//Determine index and stride
 	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	//move all of the variable definitions out of the for loop
	long xIndex, yIndex, zIndex, sphereHitIndex;
	double nearestSphereX, nearestSphereY, nearestSphereZ, distFromNearestSphereX, distFromNearestSphereY, distFromNearestSphereZ, dist;
	double xRelativeToEdge, yRelativeToEdge, zRelativeToEdge;
	int outputIndex;

	//Loop over all the energy deposition points
	for (long i = index; i < *numElements; i+=stride)
	{
		xRelativeToEdge = inputTrack[i].x-greatestSphereOffset;
		yRelativeToEdge = inputTrack[i].y-greatestSphereOffset;
		zRelativeToEdge = inputTrack[i].z-greatestSphereOffset;

		//Convert position to index in the grid of spheres
		//printf("x_shifted: %f \n",x_shifted);
		xIndex = llround((xRelativeToEdge)/sphereDiameter);
		yIndex = llround((yRelativeToEdge)/sphereDiameter);
		zIndex = llround((zRelativeToEdge)/sphereDiameter);
		
		//Determine the location of the nearest sphere in the grid (with 0,0,0 being the top left sphere, different coordinate system than the ptcls are in)
		nearestSphereX = xIndex*sphereDiameter;
		nearestSphereY = yIndex*sphereDiameter;
		nearestSphereZ = zIndex*sphereDiameter;

		//Find the distance from the nearest sphere. You have to shift x_shift by gSO to get in the same coordinate system as the sphere grid
		//An aside: I feel like there is probably a way that you could define the sphere grid that might reduce the complexity of this kernel
		//Another aside: calculating in cubes would reduce complexity as well
		distFromNearestSphereX = nearestSphereX-(xRelativeToEdge);
		distFromNearestSphereY = nearestSphereY-(yRelativeToEdge); 
		distFromNearestSphereZ = nearestSphereZ-(zRelativeToEdge); 

		//Determine if inside the nearest sphere
		dist = pow(distFromNearestSphereX,2)+pow(distFromNearestSphereY,2)+pow(distFromNearestSphereZ,2);
		dist = sqrt(dist);

		if (dist <= sphereRadius)
		{
			//Determine the Index of the sphere hit
			sphereHitIndex = xIndex + yIndex*(numSpheresLinear) + zIndex*pow(numSpheresLinear,2); //Keep in mind that for the index it starts counting at zero
			
			//Atomically add to the global counter for the output array length
			outputIndex = atomicAdd(numElementsCompacted,1);

			//Write to volumeID and edepOutput
			volumeID[outputIndex] = sphereHitIndex;
			edepOutput[outputIndex] = inputTrack[i].edep/linealDenominator; //this should be ev/nm which is same a kev/um
			//printf("volumeID: %ld %ld %ld edep: %f. \n",xIndex,yIndex,zIndex,edepOutput[i]);
		}
	}
}*/