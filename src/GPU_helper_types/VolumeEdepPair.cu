#include "VolumeEdepPair.cuh"
#include <iostream>

VolumeEdepPair::VolumeEdepPair() {}

void VolumeEdepPair::Allocate(uint64_t numInputElements)
{
	hipMalloc(&volume,numInputElements*sizeof(uint64_t));
	hipMalloc(&edep,numInputElements*sizeof(double));
	hipMallocManaged(&numElements,sizeof(int));

	//Set number of elements from number of elements allocated
	*numElements = numInputElements;
	//Zero the volume edep pairs
	hipMemset(volume,0,numInputElements*sizeof(uint64_t));
	hipMemset(edep,0,numInputElements*sizeof(double));
}

void VolumeEdepPair::Free()
{
	hipFree(volume);
	hipFree(edep);
	hipFree(numElements);
}



