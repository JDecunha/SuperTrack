#include "hip/hip_runtime.h"
//SuperTrack
#include "SimulationMethod.hh"
#include "Track.cuh"
#include "VolumeEdepPair.cuh"
//CUDA Libraries
#include <hip/hip_runtime.h>
#include <hiprand.h>


SimulationMethod::SimulationMethod(const INIReader& macroReader)
{
	_macroReader = macroReader;
}

//
// Kernel definitions
//

__global__ void SimulationMethodKernel::ZeroInt(int* toZero)
{
	*toZero = 0;
}

void SimulationMethodKernel::GenerateRandomXYShift(const std::tuple<Int_t,Int_t,Int_t,TString> &input, float **randomVals, const int &nSamples, const long &random_seed)
{
	hipMalloc(randomVals,2*sizeof(float)*nSamples); 
	
	//Random number generation on GPU
	hiprandGenerator_t randGenerator;
	hiprandCreateGenerator(&randGenerator,HIPRAND_RNG_PSEUDO_DEFAULT); //consider changing this to Mersenne Twister later
	hiprandSetPseudoRandomGeneratorSeed(randGenerator,random_seed+std::get<2>(input));
	hiprandGenerateUniform(randGenerator,*randomVals,2*nSamples);
	hiprandDestroyGenerator(randGenerator);
	hipDeviceSynchronize();
}