#include "hip/hip_runtime.h"
//SuperTrack
#include "SimulationMethod.hh"
#include "Track.cuh"
#include "VolumeEdepPair.cuh"
//CUDA Libraries
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <iostream>


SimulationMethod::SimulationMethod(const INIReader& macroReader)
{
	_macroReader = macroReader;
}

void SimulationMethod::GenerateRandomXYShift(const ThreadTask &task, float** randomVals)
{
	hipMalloc(randomVals,2*sizeof(float)*task.GetNOversamples()); 
	
	//Create the random generator
	hiprandGenerator_t randGenerator;
	hiprandCreateGenerator(&randGenerator,HIPRAND_RNG_PSEUDO_DEFAULT);

	//Seed the generator
	hiprandSetPseudoRandomGeneratorSeed(randGenerator,task.GetRandomSeed());

	//Make random numbers, and then destroy the generator
	hiprandGenerateUniform(randGenerator,*randomVals,2*task.GetNOversamples());
	hiprandDestroyGenerator(randGenerator);
	hipDeviceSynchronize();
}

//
// Kernel definitions
//

__global__ void SimulationMethodKernel::ZeroInt(int* toZero)
{
	*toZero = 0;
}


