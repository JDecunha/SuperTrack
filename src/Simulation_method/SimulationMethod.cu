#include "hip/hip_runtime.h"
//SuperTrack
#include "SimulationMethod.hh"
#include "Track.cuh"
#include "VolumeEdepPair.cuh"
//CUDA Libraries
#include <hip/hip_runtime.h>
#include <hiprand.h>


SimulationMethod::SimulationMethod(const INIReader& macroReader)
{
	_macroReader = macroReader;
}

void SimulationMethod::GenerateRandomXYShift(const ThreadTask &task, float** randomVals)
{
	hipMalloc(randomVals,2*sizeof(float)*task.GetNOversamples()); 
	
	//Random number generation on GPU
	hiprandGenerator_t randGenerator;
	hiprandCreateGenerator(&randGenerator,HIPRAND_RNG_PSEUDO_DEFAULT);
	//TODO: Change this to collate the numbers rather than add the threadID to randomSeed
	hiprandSetPseudoRandomGeneratorSeed(randGenerator,task.GetRandomSeed()+task.GetThreadID());
	hiprandGenerateUniform(randGenerator,*randomVals,2*task.GetNOversamples());
	hiprandDestroyGenerator(randGenerator);
	hipDeviceSynchronize();
}

//
// Kernel definitions
//

__global__ void SimulationMethodKernel::ZeroInt(int* toZero)
{
	*toZero = 0;
}


