#include "hip/hip_runtime.h"
//SuperTrack
#include "SimulationMethod.hh"
#include "Track.cuh"
#include "VolumeEdepPair.cuh"
//CUDA Libraries
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <iostream>


SimulationMethod::SimulationMethod(const INIReader& macroReader)
{
	_macroReader = macroReader;
}

//
// Kernel definitions
//

__global__ void SimulationMethodKernel::ZeroInt(int* toZero)
{
	*toZero = 0;
}


