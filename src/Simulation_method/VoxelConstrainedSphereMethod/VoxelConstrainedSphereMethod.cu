#include "hip/hip_runtime.h"
#include "VoxelConstrainedSphereMethod.hh"
#include "VolumeEdepPair.cuh"
#include "Track.cuh"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <iostream>

//Constructor
VoxelConstrainedSphereMethod::VoxelConstrainedSphereMethod(const INIReader& macroReader) : SimulationMethod(macroReader), _sphericalGeometry(macroReader)
{
	ParseInput();
}

//ParseInput takes the INIReader to initialize the class
void VoxelConstrainedSphereMethod::ParseInput() //Many of the inputs are currently handled by the SphericalGeometry helper struct, look there as well
{
	//Pull the number of SMs 
	int deviceId;
	hipGetDevice(&deviceId);
	hipDeviceProp_t props;
	hipGetDeviceProperties(&props, deviceId);

	//Set default number of Blocks and Threads
	int defaultNumBlocks = props.multiProcessorCount*10;
	int defaultNumThreads = 256;

	_suggestedCudaBlocks = _macroReader.GetReal("VoxelConstrainedSphere","SuggestedCudaBlocks",defaultNumBlocks);
	_suggestedCudaThreads = _macroReader.GetReal("VoxelConstrainedSphere","SuggestedCudaThreads",defaultNumThreads);
}

//Static method that the SimulationMethodFactory uses to build this simulation method
SimulationMethod* VoxelConstrainedSphereMethod::Construct(const INIReader& macroReader)
{
	return new VoxelConstrainedSphereMethod(macroReader);
}

//Called to allocate memory at the start of processing a track
void VoxelConstrainedSphereMethod::AllocateTrackProcess(Track track, ThreadTask task) 
{ 
	_oversampleIterationNumber = 0;
	_nSteps = task.GetExitPoint() - task.GetEntryPoint();

	//Allocate GPU only memory and fill with random numbers
	SimulationMethod::GenerateRandomXYShift(task, &_randomVals); 

	//Allocate memory for the track after being randomly shifted
	_randomlyShiftedTrack.AllocateEmptyTrack(_nSteps);

	//Allocate memory for the number of steps within the bounding box
	hipMallocManaged(&_numInVoxel,sizeof(int)); 

	//Allocate memory to store the StepIDs of the steps within spheres
	hipMalloc(&_inSphereTrackId,_nSteps*sizeof(int));
}

//Called repeatedly for each track oversample
void VoxelConstrainedSphereMethod::ProcessTrack(Track track, VolumeEdepPair& edepsInTarget)
{ 
	//New track. Zero values
	SimulationMethodKernel::ZeroInt<<<1,1>>>(_numInVoxel);
	SimulationMethodKernel::ZeroInt<<<1,1>>>(edepsInTarget.numElements);

	//Filter and score the tracks
		//Filter the tracks that are in the scoring voxel (box)
	VoxelConstrainedSphereMethodKernel::FilterInScoringBox<<<_suggestedCudaBlocks,_suggestedCudaThreads>>>(_sphericalGeometry,_randomVals,track,_randomlyShiftedTrack,_nSteps,_numInVoxel,_oversampleIterationNumber);	
		//Filter the tracks that land inside of a scoring sphere
	VoxelConstrainedSphereMethodKernel::FilterTrackInSphere<<<_suggestedCudaBlocks,_suggestedCudaThreads>>>(_sphericalGeometry,_randomlyShiftedTrack,_numInVoxel,edepsInTarget.numElements,_inSphereTrackId); 
		//Score the the tracks which reside in a sphere
	VoxelConstrainedSphereMethodKernel::ScoreTrackInSphere<<<_suggestedCudaBlocks,_suggestedCudaThreads>>>(_sphericalGeometry,_randomlyShiftedTrack,edepsInTarget.numElements,_inSphereTrackId,edepsInTarget);

	_oversampleIterationNumber++;
}

//Called at the end of processing a track
void VoxelConstrainedSphereMethod::FreeTrackProcess()
{ 
	//Free directly allocated memory
	hipFree(_inSphereTrackId);
	hipFree(_randomVals);
	hipFree(_numInVoxel);

	//Free my classes
	_randomlyShiftedTrack.Free();
}

//Called at the end of processing all of the tracks
void VoxelConstrainedSphereMethod::Free()
{ 

}

//
//Kernel definitions
//

__global__ void VoxelConstrainedSphereMethodKernel::FilterInScoringBox(SphericalGeometry geometry, float* randomVals, Track inputTrack, Track outputTrack, int numElements, int *numElementsCompacted, int oversampleIterationNumber)
{
	//This function
	//1.) Applies the random shift to the x,y coordinates
	//2.) Checks which edep events are in the box
	//3.) Performs stream compaction on those events which are in the box
	//We are using stream compaction to avoid a monolithic kernel with large blocks within if-statements which reduces warp efficiency

	//Put definitions outside of for-loop to prevent repeat constructor calls
	double x_shifted; double y_shifted; int outputIndex;

	//Determine index and strid
    int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	//Counters for shared memory atomics
	int localPosition;
	__shared__ int localIndexCounter;
	
	//Convert random shifts in to appropriate range
	double x_shift = ((randomVals[(oversampleIterationNumber*2)]*geometry.greatestSphereOffset*2)-geometry.greatestSphereOffset);
	double y_shift = ((randomVals[(oversampleIterationNumber*2+1)]*geometry.greatestSphereOffset*2)-geometry.greatestSphereOffset);

	//The value we compare to, to check if it's in the box
	double box_edge = abs(geometry.greatestSphereOffset)+(geometry.sphereRadius);

	//Loop over all the energy deposition points
	for (int i = index; i < numElements; i+=stride)
	{
		//Apply random shift
		x_shifted = inputTrack.x[i] + x_shift;
		y_shifted = inputTrack.y[i] + y_shift;

		//Set local position to negative value, only takes on positive value if predicate is true
		localPosition = -1;

		//Zero the local counter
		if (threadIdx.x == 0) 
		{
			localIndexCounter = 0;
		}
		__syncthreads();

		//Check if in box, if true assign the local index position
		//we don't have to check Z, the tracks are generated so they are never outside in Z
		if (abs(x_shifted) < box_edge  && abs(y_shifted) < box_edge) 
		{
			localPosition = atomicAdd(&localIndexCounter,1);
		}
		__syncthreads();

		//Add the local counter to the global counter
		if (threadIdx.x == 0)
		{
			localIndexCounter = atomicAdd(numElementsCompacted,localIndexCounter);
		}
		__syncthreads();

		//If predicate is true, then write the track to position localCounter+localPosition (localCounter now stores the globalCounter value because of the atomic add)
		if(localPosition != -1)
		{
			//Atomically add to the global counter for the output array length
			outputIndex = localPosition+localIndexCounter;

			//Copy the track inside the box over to the new array
			outputTrack.x[outputIndex] = x_shifted;
			outputTrack.y[outputIndex] = y_shifted;
			outputTrack.z[outputIndex] = inputTrack.z[i];
			outputTrack.edep[outputIndex] = inputTrack.edep[i];
		}
		__syncthreads();
	}
}

__global__ void VoxelConstrainedSphereMethodKernel::FilterTrackInSphere(SphericalGeometry geometry, Track inputTrack, int *numElements, int *numElementsCompacted, int *trackIdInSphere)
{

	//printf("%f %f \n",geometry.sphereDiameter,geometry.scoringRegionHalfLength);

	//move all of the variable definitions out of the for loop
	double distFromNearestSphereX, distFromNearestSphereY, distFromNearestSphereZ, dist;

	//Determine index and stride
 	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	//Counters for shared memory atomics
	int localPosition;
	__shared__ int localIndexCounter;

	//Pre-calculate values
	double sphereDiameter = geometry.sphereDiameter; 
	double sphereRadiusMag = geometry.sphereRadius*geometry.sphereRadius; 

	//Loop over all the energy deposition points
	for (long i = index; i < *numElements; i+=stride)
	{		
		//Find distance to the nearest sphere.
		//For performance reasons, we work in an arbitrary coordinate system here, rather than the "global" coordinate system
		//The "global" coordinate systrem is relative to the greatest sphere offset
		//In the later scoring kernel we work in the global coordinate system, and that's why we subtract the greatest sphere offset there
		distFromNearestSphereX = llrint((inputTrack.x[i])/sphereDiameter)*geometry.sphereDiameter-(inputTrack.x[i]);
		distFromNearestSphereY = llrint((inputTrack.y[i])/sphereDiameter)*geometry.sphereDiameter-(inputTrack.y[i]); 
		distFromNearestSphereZ = llrint((inputTrack.z[i])/sphereDiameter)*geometry.sphereDiameter-(inputTrack.z[i]); 

		//Determine if inside the nearest sphere
		dist = (distFromNearestSphereX*distFromNearestSphereX)+(distFromNearestSphereY*distFromNearestSphereY)+(distFromNearestSphereZ*distFromNearestSphereZ);

		//Set local position to negative value, only takes on positive value if predicate is true
		localPosition = -1;

		//Zero the local counter
		if (threadIdx.x == 0) 
		{
			localIndexCounter = 0;
		}
		__syncthreads();

		//Check if in sphere, then assign local index position
		if (dist <= sphereRadiusMag)
		{
			localPosition = atomicAdd(&localIndexCounter,1);
		}
		__syncthreads();

		//Add the local counter to the global counter
		if (threadIdx.x == 0)
		{
			localIndexCounter = atomicAdd(numElementsCompacted,localIndexCounter);
		}
		__syncthreads();

		//If predicate is true, then write the track to position localCounter+localPosition (localCounter now stores the globalCounter value because of the atomic add)
		if (localPosition != -1)
		{
			//Atomically add to the global counter for the output array length
			trackIdInSphere[localPosition+localIndexCounter] = i;
		}
		__syncthreads();

	}
}

__global__ void VoxelConstrainedSphereMethodKernel::ScoreTrackInSphere(SphericalGeometry geometry, Track inputTrack, int *numElements, int *trackIdInSphere, VolumeEdepPair outputPair)
{
	//move all of the variable definitions out of the for loop
	long xIndex, yIndex, zIndex, sphereHitIndex;

	//Pre-calculate some values
	double sphereDiameter = geometry.sphereDiameter;
	double linealDenominator = (2./3.)*sphereDiameter; 

	//Determine index and stride
 	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	//Loop over all the energy deposition points
	for (uint64_t i = index; i < *numElements; i+=stride)
	{		
		xIndex = llrint((inputTrack.x[trackIdInSphere[i]]-geometry.greatestSphereOffset)/sphereDiameter);
		yIndex = llrint((inputTrack.y[trackIdInSphere[i]]-geometry.greatestSphereOffset)/sphereDiameter);
		zIndex = llrint((inputTrack.z[trackIdInSphere[i]]-geometry.greatestSphereOffset)/sphereDiameter);

		//Determine the Index of the sphere hit
		sphereHitIndex = xIndex + yIndex*geometry.numSpheresLinear+ zIndex*geometry.numSpheresLinear*geometry.numSpheresLinear; //Keep in mind that for the index it starts counting at zero

		//Write to volumeID and edepOutput
		outputPair.volume[i] = sphereHitIndex;
		outputPair.edep[i] = inputTrack.edep[trackIdInSphere[i]]/linealDenominator; //this should be ev/nm which is same a kev/um
	}
}

