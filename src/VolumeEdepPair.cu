#include "VolumeEdepPair.cuh"
#include <iostream>

VolumeEdepPair::VolumeEdepPair() {}

void VolumeEdepPair::Allocate(uint64_t numInputElements)
{
	hipMalloc(&volume,numInputElements*sizeof(uint64_t));
	hipMalloc(&edep,numInputElements*sizeof(double));
	hipMallocManaged(&numElements,sizeof(int));

	*numElements = numInputElements;
}

void VolumeEdepPair::Free()
{
	hipFree(volume);
	hipFree(edep);
	hipFree(numElements);
}



